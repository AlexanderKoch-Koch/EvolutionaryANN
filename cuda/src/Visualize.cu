#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "Synapse.h"
#include "Hyperparameters.h"
#include "Parameters.h"
#include "TensorboardInterface.h"


__global__ void printSynapses(struct Synapse *d_synapses, size_t pitch){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    if(neuron < NUM_NEURONS){
        float weight_sum = 0;
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        for(int synapse = 0; synapse< NUM_SYNAPSES_PER_NEURON; synapse++){
            printf("neuron: %d, synapse: %d, weight: %.2f, activity: %.2f, input: %d\n",
                neuron, synapse, neuron_array[synapse].weight,
                neuron_array[synapse].activity, neuron_array[synapse].input);
            weight_sum += neuron_array[synapse].weight;
        }
        printf("avr weight: %.2f  ", weight_sum / NUM_SYNAPSES_PER_NEURON);
    }
}

__global__ void print_synapse_stats(struct Synapse *d_synapses, size_t pitch){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    if(neuron < NUM_NEURONS){
        float weight_sum = 0;
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        for(int synapse = 0; synapse< NUM_SYNAPSES_PER_NEURON; synapse++){
            weight_sum += neuron_array[synapse].weight;
        }
        float mean = weight_sum /NUM_SYNAPSES_PER_NEURON;
        printf("avr weight: %.2f  ", mean);
        //compute standard deviation
        float standard_deviation = 0.0;
        for(int synapse = 0; synapse< NUM_SYNAPSES_PER_NEURON; synapse++){
            standard_deviation += (neuron_array[synapse].weight - mean) * (neuron_array[synapse].weight - mean);
        }
        printf("standard_deviation: %.2f   ", standard_deviation);
    }
}

__global__ void printNeurons(int *d_neuron_outputs, float *d_weighted_sums){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    printf("neuron: %d, weighted sum: %.2f, output: %d\n", neuron, d_weighted_sums[neuron], d_neuron_outputs[neuron]);
}


void neuron_stats(int *d_neuron_outputs, unsigned long step){
    int *neuron_outputs = (int*) malloc(sizeof(int) * NUM_NEURONS);
    hipMemcpy(neuron_outputs, d_neuron_outputs, sizeof(int) * NUM_NEURONS, hipMemcpyDeviceToHost);
    int output_sum = 0;
    for(int i = 0; i <NUM_NEURONS; i++){
        output_sum += neuron_outputs[i];
    }
    write_scalar(step, output_sum/(float)NUM_NEURONS, "avr_output");
}

__global__ void print_parameters(struct Parameters *d_parameters){
    printf(" threshold_randomness_factor %.3f ", d_parameters->threshold_randomness_factor);   
}