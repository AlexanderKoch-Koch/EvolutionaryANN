#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "Synapse.h"
#include "Hyperparameters.h"
#include "Parameters.h"
#include <math.h>



__global__ void compute(struct Synapse *d_synapses, int *d_neuron_outputs, size_t pitch, hiprandState_t *d_curand_states, struct Parameters *d_parameters){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    if(neuron < NUM_NEURONS){
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        float weighted_sum = 0.0;
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            weighted_sum += neuron_array[synapse].input * neuron_array[synapse].weight;
        }
        float random = d_parameters->threshold_randomness_factor * hiprand_normal(&d_curand_states[neuron]);
        //printf("weighted_sum: %.2f, random_add: %.2f  ", weighted_sum, random);
        if(weighted_sum + random >= THRESHOLD){
            d_neuron_outputs[neuron] = 1;
            //printf("firing");
        }else{
            d_neuron_outputs[neuron] = 0;
        }
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            neuron_array[synapse].activity *= ACTIVITY_DISCOUNT_FACTOR;
            //neuron_array[synapse].activity += neuron_array[synapse].input * d_neuron_outputs[neuron] * neuron_array[synapse].weight;
            //neuron_array[synapse].activity += neuron_array[synapse].input * fabs(neuron_array[synapse].weight) * (d_neuron_outputs[neuron] - 0.5);
            neuron_array[synapse].activity += neuron_array[synapse].input * (d_neuron_outputs[neuron] - 0.5);
        }
    }
}


__global__ void compute_t1(struct Synapse *d_t1_synapses, int *d_t1_neuron_outputs, size_t t1_pitch, hiprandState_t *d_t1_curand_states, struct Parameters *d_parameters){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    if(neuron < NUM_T1_NEURONS){
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_t1_synapses + neuron * t1_pitch);
        float weighted_sum = 0.0;
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            weighted_sum += neuron_array[synapse].input * neuron_array[synapse].weight;
        }
        float random = d_parameters->threshold_randomness_factor * hiprand_normal(&d_t1_curand_states[neuron]);
        //printf("weighted_sum: %.2f, random_add: %.2f  ", weighted_sum, random);
        if(weighted_sum + random >= THRESHOLD){
            d_t1_neuron_outputs[neuron] = 1;
            //printf("firing");
        }else{
            d_t1_neuron_outputs[neuron] = 0;
        }
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            neuron_array[synapse].activity *= ACTIVITY_DISCOUNT_FACTOR;
            neuron_array[synapse].activity += neuron_array[synapse].input * (d_t1_neuron_outputs[neuron] - 0.5);
        }
    }
}


__global__ void read(struct Synapse *d_synapses, size_t pitch){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    if(neuron < NUM_NEURONS){
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            neuron_array[synapse].input = (*neuron_array[synapse].p_presynaptic_output);
        }
    }
}

__global__ void read_t1(struct Synapse *d_t1_synapses, size_t t1_pitch){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    if(neuron < NUM_T1_NEURONS){
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_t1_synapses + neuron * t1_pitch);
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            neuron_array[synapse].input = (*neuron_array[synapse].p_presynaptic_output);
        }
    }
}


__global__ void learn(struct Synapse *d_synapses, float learning_factor, size_t pitch, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t *d_curand_state){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    if(neuron < NUM_NEURONS){
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            //neuron_array[synapse].weight += LEARNING_RATE * reward * neuron_array[synapse].activity;
            //neuron_array[synapse].weight += LEARNING_RATE * reward * neuron_array[synapse].activity * fabs(MAX_ABS_WEIGHT - fabs(neuron_array[synapse].weight));
            neuron_array[synapse].weight += LEARNING_RATE * learning_factor * neuron_array[synapse].activity;
            neuron_array[synapse].weight *= pow(2.0, -WEIGHT_VALUE_LIMIT_FACTOR * neuron_array[synapse].weight * neuron_array[synapse].weight);

            //randomly reconnect if weight too small
            if(fabsf(neuron_array[synapse].weight) < MIN_ABS_WEIGHT){
                float new_weight = hiprand_uniform(&d_curand_state[neuron]) + MIN_START_WEIGHT;
                neuron_array[synapse].weight = new_weight;
                //printf("reconnecting");
                int rand_input = hiprand(&d_curand_state[neuron]) % (NUM_NEURONS + NUM_INPUTS);
                if(rand_input < NUM_NEURONS){
                    //connect to other neuron
                    neuron_array[synapse].p_presynaptic_output = &d_neuron_outputs[rand_input];
                }else{
                    //connect to brain input
                    neuron_array[synapse].p_presynaptic_output = &d_brain_inputs[rand_input - NUM_NEURONS];
                }
            }
        }
    }
}


__global__ void learn_t1(struct Synapse *d_t1_synapses, float reward, size_t t1_pitch, int *d_t1_neuron_outputs, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t *d_t1_curand_states){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(neuron < NUM_T1_NEURONS){
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_t1_synapses + neuron * t1_pitch);
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            //neuron_array[synapse].weight += LEARNING_RATE * reward * neuron_array[synapse].activity;
            //neuron_array[synapse].weight += LEARNING_RATE * reward * neuron_array[synapse].activity * fabs(MAX_ABS_WEIGHT - fabs(neuron_array[synapse].weight));
            neuron_array[synapse].weight += LEARNING_RATE * reward * neuron_array[synapse].activity;
            neuron_array[synapse].weight *= pow(2.0, -WEIGHT_VALUE_LIMIT_FACTOR * neuron_array[synapse].weight * neuron_array[synapse].weight);

            //randomly reconnect if weight too small
            if(fabsf(neuron_array[synapse].weight) < MIN_ABS_WEIGHT){
                int rand_input = hiprand(&d_t1_curand_states[neuron]) % (NUM_NEURONS + NUM_T1_NEURONS + NUM_INPUTS);
                if(rand_input < NUM_NEURONS){
                    //connect to neuron from main ANN
                    neuron_array[synapse].p_presynaptic_output = &d_neuron_outputs[rand_input];
                }else if(rand_input < NUM_NEURONS + NUM_T1_NEURONS){
                    //connect to T1 neuron
                    neuron_array[synapse].p_presynaptic_output = &d_t1_neuron_outputs[rand_input - NUM_NEURONS];
                }else{
                    //connect to brain input
                    neuron_array[synapse].p_presynaptic_output = &d_brain_inputs[rand_input - (NUM_NEURONS + NUM_T1_NEURONS)];
                }
            }
        }
    }
}



__global__ void reset_synapses(struct Synapse *d_synapses, float *d_weighted_sums, size_t pitch){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;

    if(neuron < NUM_NEURONS){
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            neuron_array[synapse].input = 0;
            neuron_array[synapse].activity = 0;
        }
    }
}


__global__ void update_parameters(struct Parameters *d_parameters){
    d_parameters->threshold_randomness_factor *= 1;
    //printf("new factor %.2f ", d_parameters->threshold_randomness_factor);
}