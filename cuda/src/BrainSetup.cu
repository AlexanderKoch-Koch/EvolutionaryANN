#include "hip/hip_runtime.h"
#include "BrainSetup.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "Synapse.h"
#include "Hyperparameters.h"


__global__ void init_random_seed(unsigned int seed, hiprandState_t *d_curand_state) {
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    hiprand_init(seed, neuron, 0, &d_curand_state[neuron]);
}


__global__ void init_synapses(struct Synapse *d_synapses, size_t pitch, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t *d_curand_state){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    
    if(neuron < NUM_NEURONS){
        //set randomly 3/13 of the outputs to 1
        d_neuron_outputs[neuron] = hiprand(&d_curand_state[neuron]) % 13 / 10;

        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            //random weight between MIN_START_WEIGHT and MIN_START_WEIGHT + 1
            float new_weight = hiprand_uniform(&d_curand_state[neuron]) + MIN_START_WEIGHT;
            neuron_array[synapse].weight = new_weight;
            neuron_array[synapse].activity = 0.0;
            //printf("neuron %d synapse: %d weight %.2f activity: %.2f\n", neuron, synapse, neuron_array[synapse].weight, neuron_array[synapse].activity);
            //random input
            int rand_input = hiprand(&d_curand_state[neuron]) % (NUM_NEURONS + NUM_INPUTS);
            if(rand_input < NUM_NEURONS){
                //connect to other neuron
                neuron_array[synapse].p_presynaptic_output = &d_neuron_outputs[rand_input];
            }else{
                //connect to brain input
                neuron_array[synapse].p_presynaptic_output = &d_brain_inputs[rand_input - NUM_NEURONS];
            }
        }
    }
}


__global__ void init_t1_synapses(struct Synapse *d_t1_synapses, size_t t1_pitch, int *d_t1_neuron_outputs, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t *d_t1_curand_states){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    
    if(neuron < NUM_T1_NEURONS){
        //set randomly 3/13 of the outputs to 1
        d_t1_neuron_outputs[neuron] = hiprand(&d_t1_curand_states[neuron]) % 13 / 10;

        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_t1_synapses + neuron * t1_pitch);
        
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            //random weight between MIN_START_WEIGHT and MIN_START_WEIGHT + 1
            float new_weight = hiprand_uniform(&d_t1_curand_states[neuron]) + MIN_START_WEIGHT;
            neuron_array[synapse].weight = new_weight;
            neuron_array[synapse].activity = 0.0;
            //printf("neuron %d synapse: %d weight %.2f activity: %.2f\n", neuron, synapse, neuron_array[synapse].weight, neuron_array[synapse].activity);
            //random input
            int rand_input = hiprand(&d_t1_curand_states[neuron]) % (NUM_NEURONS + NUM_T1_NEURONS + NUM_INPUTS);
            if(rand_input < NUM_NEURONS){
                //connect to neuron from main ANN
                neuron_array[synapse].p_presynaptic_output = &d_neuron_outputs[rand_input];
                //printf("connect to neuron from main ANN");
            }else if(rand_input < NUM_NEURONS + NUM_T1_NEURONS){
                //connect to T1 neuron
                neuron_array[synapse].p_presynaptic_output = &d_t1_neuron_outputs[rand_input - NUM_NEURONS];
                //printf("connect to T1 neuron");
            }else{
                //connect to brain input
                neuron_array[synapse].p_presynaptic_output = &d_brain_inputs[rand_input - (NUM_NEURONS + NUM_T1_NEURONS)];
                //printf("connect to brain input");
            }
        }
    }
}
