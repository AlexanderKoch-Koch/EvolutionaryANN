#include "hip/hip_runtime.h"
#include "BrainSetup.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "Synapse.h"
#include "Hyperparameters.h"


__global__ void init_random_seed(unsigned int seed, hiprandState_t *d_curand_state) {
    hiprand_init(seed, blockIdx.x, 0, d_curand_state);
}


__global__ void init_synapses(struct Synapse *d_synapses, size_t pitch, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t *d_curand_state){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    
    if(neuron < NUM_NEURONS){
        skipahead(neuron, d_curand_state);
        //set randomly 3/13 of the outputs to 1
        d_neuron_outputs[neuron] = hiprand(d_curand_state) % 13 / 10;

        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            //random weight between MIN_START_WEIGHT and MIN_START_WEIGHT + 1
            float new_weight = hiprand_uniform(d_curand_state) + MIN_START_WEIGHT;
            //printf("neuron %d synapse: %d weight %.2f\n", neuron, synapse, new_weight);
            neuron_array[synapse].weight = new_weight;
            //random input
            int rand_input = hiprand(d_curand_state) % (NUM_NEURONS + NUM_INPUTS);
            if(rand_input < NUM_NEURONS){
                //connect to other neuron
                neuron_array[synapse].p_presynaptic_output = &d_neuron_outputs[rand_input];
            }else{
                //connect to brain input
                neuron_array[synapse].p_presynaptic_output = &d_brain_inputs[rand_input - NUM_NEURONS];
            }
        }
    }
}
