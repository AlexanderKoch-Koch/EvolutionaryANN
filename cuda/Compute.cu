#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "Synapse.h"
#include "Hyperparameters.h"


__global__ void compute_synapses(struct Synapse *d_synapses, float *d_weighted_sums, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    float sum = neuron_array[synapse].weight *  (*neuron_array[synapse].p_presynaptic_output);
    atomicAdd(&d_weighted_sums[neuron], sum);
    //printf("neuron: %d, synapse: %d,  adding %d\n", neuron, synapse, *neuron_array[synapse].p_presynaptic_output);
}


__global__ void compute_neurons(int *d_neuron_outputs, float *d_weighted_sums){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(d_weighted_sums[neuron] >= THRESHOLD){
        d_neuron_outputs[neuron] = 1;
    }else{
        d_neuron_outputs[neuron] = 0;
    }
    
    //reset weighted sum
    d_weighted_sums[neuron] = 0.0;
}


__global__ void tag_synapses(struct Synapse *d_synapses, int *d_neuron_outputs, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    neuron_array[synapse].activity *= ACTIVITY_DISCOUNT_FACTOR;
    neuron_array[synapse].activity += neuron_array[synapse].input * d_neuron_outputs[neuron] * neuron_array[synapse].weight;
}


__global__ void learn(struct Synapse *d_synapses, float reward, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    
    neuron_array[synapse].weight += LEARNING_RATE * reward * neuron_array[synapse].activity;
}


__global__ void reset_synapses(struct Synapse *d_synapses, float *d_weighted_sums, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    
    neuron_array[synapse].input = 0;
    neuron_array[synapse].activity = 0;
    d_weighted_sums[neuron] = 0;
}