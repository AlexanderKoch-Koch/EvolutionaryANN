#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_NEURONS 10
#define NUM_SYNAPSES_PER_NEURON 3

struct Synapse{
  float weight;
  float activity;
  int input;
  int *p_presynaptic_output;
};

__global__ void init_random_seed(unsigned int seed, hiprandState_t d_curand_state) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &d_curand_state);
}

__global__ void init(struct Synapse *d_synapses, size_t pitch, hiprandState_t d_curand_state){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    skipahead(synapse * (neuron + 1), &d_curand_state);
    float new_weight = hiprand_uniform(&d_curand_state);
    
    struct Synapse *row_a = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    row_a[synapse].weight = new_weight;
    
    printf("neuron: %d, synapse: %d, new_weight: %.2f\n", neuron, synapse,  new_weight);
}

__global__ void compute(struct Synapse *d_synapses, int *d_neuron_outputs, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *row_a = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
}

__global__ void printSynapses(struct Synapse *d_synapses, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *row_a = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    printf("neuron: %d, synapse: %d, weight: %.2f, activity: %.2f, input: %d\n", neuron, synapse, row_a[synapse].weight, row_a[synapse].activity, row_a[synapse].input);
}

__global__ void printNeurons(int *d_neuron_outputs){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    printf("neuron: %d, output: %d\n", neuron, d_neuron_outputs[neuron]);
}

int main(void){
    hiprandState_t d_curand_state;
    hipMalloc((void**) &d_curand_state, sizeof(hiprandState_t));
    init_random_seed<<<1, 1>>>(time(0), d_curand_state);
  
    size_t dev_pitch;
    struct Synapse *d_synapses;
    int *d_neuron_outputs;
    hipMalloc(&d_neuron_outputs, sizeof(int) * NUM_NEURONS);
    hipMallocPitch(&d_synapses, &dev_pitch, NUM_SYNAPSES_PER_NEURON * sizeof(struct Synapse), NUM_NEURONS);
    
    dim3 synapses_dim(NUM_SYNAPSES_PER_NEURON, NUM_NEURONS, 1);

    init<<<1, synapses_dim>>>(d_synapses, dev_pitch, d_curand_state);
    hipDeviceSynchronize();
    //compute<<<1, synapses_dim>>>(d_synapses, d_neuron_outputs, dev_pitch);
    hipDeviceSynchronize();
    printSynapses<<<1, synapses_dim>>>(d_synapses, dev_pitch);
    printNeurons<<<1, NUM_NEURONS>>>(d_neuron_outputs);
    hipDeviceSynchronize();
    return 0;
}