#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_NEURONS 1000
#define NUM_INPUTS 4
#define NUM_OUTPUTS 3
#define NUM_SYNAPSES_PER_NEURON 1000
#define THRESHOLD 0

struct Synapse{
  float weight;
  float activity;
  int input;
  int *p_presynaptic_output;
};

__global__ void init_random_seed(unsigned int seed, hiprandState_t d_curand_state) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &d_curand_state);
}

__global__ void init_synapses(struct Synapse *d_synapses, size_t pitch, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t d_curand_state){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    skipahead(synapse * (neuron + 1), &d_curand_state);
    float new_weight = hiprand_uniform(&d_curand_state);
    
    struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    neuron_array[synapse].weight = new_weight;
    
    int rand_input = hiprand(&d_curand_state) % (NUM_NEURONS + NUM_INPUTS);
    if(rand_input < NUM_NEURONS){
        //connect to other neuron
        neuron_array[synapse].p_presynaptic_output = &d_neuron_outputs[rand_input];
    }else{
        //connect to brain input
        neuron_array[synapse].p_presynaptic_output = &d_brain_inputs[rand_input - NUM_NEURONS];
    }
    //printf("neuron: %d, synapse: %d, new_weight: %.2f\n", neuron, synapse,  new_weight);
}

__global__ void init_neurons(int *d_neuron_outputs, float *d_weighted_sums){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    
    d_neuron_outputs[neuron] = 0;
    d_weighted_sums[neuron] = 0;
}

__global__ void compute_synapses(struct Synapse *d_synapses, float *d_weighted_sums, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    float sum = neuron_array[synapse].weight *  (*neuron_array[synapse].p_presynaptic_output);
    atomicAdd(&d_weighted_sums[neuron], sum);
    //printf("neuron: %d, synapse: %d,  adding %d\n", neuron, synapse, *neuron_array[synapse].p_presynaptic_output);
}

__global__ void compute_neurons(int *d_neuron_outputs, float *d_weighted_sums){
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(d_weighted_sums[neuron] >= THRESHOLD){
        d_neuron_outputs[neuron] = 1;
        printf("firing :)");
    }else{
        d_neuron_outputs[neuron] = 0;
    }
    
    //reset weighted sum
    d_weighted_sums[neuron] = 0.0;
}

__global__ void printSynapses(struct Synapse *d_synapses, size_t pitch){
    int synapse = blockIdx.x*blockDim.x + threadIdx.x;
    int neuron = blockIdx.y*blockDim.y + threadIdx.y;

    struct Synapse *row_a = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
    printf("neuron: %d, synapse: %d, weight: %.2f, activity: %.2f, input: %d\n", neuron, synapse, row_a[synapse].weight, row_a[synapse].activity, row_a[synapse].input);
}

__global__ void printNeurons(int *d_neuron_outputs, float *d_weighted_sums){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    printf("neuron: %d, weighted sum: %.2f, output: %d\n", neuron, d_weighted_sums[neuron], d_neuron_outputs[neuron]);
}

int main(void){
    //initialize hiprand
    hiprandState_t d_curand_state;
    hipMalloc((void**) &d_curand_state, sizeof(hiprandState_t));
    init_random_seed<<<1, 1>>>(time(0), d_curand_state);
    
    //allocate memory for neurons
    size_t dev_pitch;
    struct Synapse *d_synapses;
    int *d_neuron_outputs;
    float *d_weighted_sums;
    int *d_brain_inputs;
    hipMalloc(&d_brain_inputs, sizeof(int) * NUM_INPUTS);
    hipMalloc(&d_weighted_sums, sizeof(float) * NUM_NEURONS);
    hipMalloc(&d_neuron_outputs, sizeof(int) * NUM_NEURONS);
    hipMallocPitch(&d_synapses, &dev_pitch, NUM_SYNAPSES_PER_NEURON * sizeof(struct Synapse), NUM_NEURONS);
    
    dim3 synapses_dim(NUM_SYNAPSES_PER_NEURON, NUM_NEURONS, 1);

    // initialize brain
    init_synapses<<<1, synapses_dim>>>(d_synapses, dev_pitch, d_neuron_outputs, d_brain_inputs, d_curand_state);
    init_neurons<<<1, NUM_NEURONS>>>(d_neuron_outputs, d_weighted_sums);
    hipDeviceSynchronize();
    
    //set brain inputs
    int inputs[] = {1, 0, 1, 1};
    hipMemcpy(d_brain_inputs, inputs, sizeof(int) * NUM_INPUTS, hipMemcpyHostToDevice);
    
    compute_synapses<<<1, synapses_dim>>>(d_synapses, d_weighted_sums, dev_pitch);
    hipDeviceSynchronize();
    
    compute_neurons<<<1, NUM_NEURONS>>>(d_neuron_outputs, d_weighted_sums);
    hipDeviceSynchronize();
    
    //printSynapses<<<1, synapses_dim>>>(d_synapses, dev_pitch);
    //printNeurons<<<1, NUM_NEURONS>>>(d_neuron_outputs, d_weighted_sums);
    hipDeviceSynchronize();
    
    return 0;
}