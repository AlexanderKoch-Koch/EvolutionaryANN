#include "hip/hip_runtime.h"
#include "BrainSetup.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "Synapse.h"
#include "Hyperparameters.h"


__global__ void init_random_seed(unsigned int seed, hiprandState_t *d_curand_state) {

    /* we have to initialize the state */
    hiprand_init(seed, blockIdx.x, 0, d_curand_state);
}


__global__ void init_synapses(struct Synapse *d_synapses, size_t pitch, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t *d_curand_state){
    int neuron = blockIdx.x*blockDim.x + threadIdx.x;
    if(neuron < NUM_NEURONS){

        d_neuron_outputs[neuron] = hiprand(d_curand_state) % 2;
    
        struct Synapse *neuron_array = (struct Synapse *) ((char*)d_synapses + neuron * pitch);
        skipahead(neuron, d_curand_state);
        
        for(int synapse = 0; synapse < NUM_SYNAPSES_PER_NEURON; synapse++){
            create_synapse(&neuron_array[synapse], d_neuron_outputs, d_brain_inputs, d_curand_state);
        }
    }
}

__device__ void create_synapse(struct Synapse *d_synapse, int *d_neuron_outputs, int *d_brain_inputs, hiprandState_t *d_curand_state){
    float new_weight = hiprand_uniform(d_curand_state) + MIN_START_WEIGHT;
            
    d_synapse->weight = new_weight;
            
    int rand_input = hiprand(d_curand_state) % (NUM_NEURONS + NUM_INPUTS);
    if(rand_input < NUM_NEURONS){
        //connect to other neuron
        d_synapse->p_presynaptic_output = &d_neuron_outputs[rand_input];
    }else{
        //connect to brain input
        d_synapse->p_presynaptic_output = &d_brain_inputs[rand_input - NUM_NEURONS];
    }
}
