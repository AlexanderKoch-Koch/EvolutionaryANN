#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "Synapse.h"
#include "BrainSetup.h"
#include "Visualize.h"
#include "Compute.h"
#include "Hyperparameters.h"

dim3 synapses_dim(NUM_SYNAPSES_PER_NEURON, NUM_NEURONS, 1);
size_t dev_pitch;
struct Synapse *d_synapses;
int *d_neuron_outputs;
float *d_weighted_sums;
int *d_brain_inputs;
hiprandState_t *d_curand_state;

void init(){
    hipMalloc(&d_curand_state, sizeof(hiprandState_t));
    init_random_seed<<<1, 1>>>(1, d_curand_state);
    //allocate memory on the device
    hipMalloc(&d_brain_inputs, sizeof(int) * NUM_INPUTS);
    hipMalloc(&d_weighted_sums, sizeof(float) * NUM_NEURONS);
    hipMalloc(&d_neuron_outputs, sizeof(int) * NUM_NEURONS);
    hipMallocPitch(&d_synapses, &dev_pitch, NUM_SYNAPSES_PER_NEURON * sizeof(struct Synapse), NUM_NEURONS);
    
    // initialize brain
    init_synapses<<<1, synapses_dim>>>(d_synapses, dev_pitch, d_neuron_outputs, d_brain_inputs, d_curand_state);
    init_neurons<<<1, NUM_NEURONS>>>(d_neuron_outputs, d_weighted_sums);
    hipDeviceSynchronize();
    
    printSynapses<<<1, synapses_dim>>>(d_synapses, dev_pitch);
    printNeurons<<<1, NUM_NEURONS>>>(d_neuron_outputs, d_weighted_sums);
    hipDeviceSynchronize();
}


int* think(int *inputs){
    //set brain inputs
    hipMemcpy(d_brain_inputs, inputs, sizeof(int) * NUM_INPUTS, hipMemcpyHostToDevice);
    
    //sum up the inputs
    compute_synapses<<<1, synapses_dim>>>(d_synapses, d_weighted_sums, dev_pitch);
    hipDeviceSynchronize();
    
    //decide Threshold
    compute_neurons<<<1, NUM_NEURONS>>>(d_neuron_outputs, d_weighted_sums);
    hipDeviceSynchronize();
    tag_synapses<<<1, synapses_dim>>>(d_synapses, d_neuron_outputs, dev_pitch);
    //copy results back to host
    int *outputs = (int*) malloc(sizeof(int) * NUM_OUTPUTS);
    hipMemcpy(outputs, d_neuron_outputs, sizeof(int) * NUM_OUTPUTS, hipMemcpyDeviceToHost);
    return outputs;
}

void process_reward(float reward){
    printf("reward is %.2f", reward);
}

void reset_memory(){
    reset_synapses<<<1, synapses_dim>>>(d_synapses, d_weighted_sums, dev_pitch);
}

void release_memory(){
    hipFree(d_brain_inputs);
    hipFree(d_neuron_outputs);
    hipFree(d_synapses);
    hipFree(d_weighted_sums);
}